
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

//
// General
//

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

// Reduction kernel for sum
__global__ void reduction_kernel(double* sum, const double* a, long N) {
	__shared__ double smem[BLOCK_SIZE];
	int idx = (blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx < N) smem[threadIdx.x] = a[idx];
	else smem[threadIdx.x] = 0;

	__syncthreads();
	if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
	__syncthreads();
	if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
	__syncthreads();
	if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
	__syncthreads();
	if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x + 64];
	__syncthreads();
	if (threadIdx.x <  32) {
		smem[threadIdx.x] += smem[threadIdx.x + 32];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 16];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 8];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 4];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 2];
		__syncwarp();
		if (threadIdx.x == 0) {
			sum[blockIdx.x] = smem[0] + smem[1];
		}
	}
}

// CPU matrix-vector multiplication implementation
// Assume A is square and is stored in row major: A[i, j] = A[N * i + j]
void matrix_vector_ref(double* Ax_ref, const double* A, const double* x, long N) {
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) {
    for (long j = 0; j < N; j++) {
      Ax_ref[i] += A[N * i + j] * x[j];
    }
  }
}

// Kernel for matrix-vector product per result entry
__global__ void matrix_vector_kernel(double* Ax, const double* A, const double* x, long N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    Ax[idx] = 0;
    for (long j = 0; j < N; j++) {
      Ax[idx] += A[idx * N + j] * x[j];
    }
  }
}

int main() {
  long N = (1UL<<10);

  double *A, *x, *A_d, *x_d, *Ax_d, *Ax_ref, *Ax;

  // Initialize vector and matrix
  hipHostMalloc((void**)&A, N * N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = drand48();
    for (long j = 0; j < N; j++) {
      A[i * N + j] = drand48();
    }
  }

  // Get reference product
  hipHostMalloc((void**)&Ax_ref, N * sizeof(double), hipHostMallocDefault);
  double tt = omp_get_wtime();
  matrix_vector_ref(Ax_ref, A, x, N);
  printf("CPU Bandwidth = %f GB/s\n", 2*N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  // Get GPU product
  hipMalloc(&A_d, N*N*sizeof(double));
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&Ax_d, N*sizeof(double));
  hipHostMalloc(&Ax, N*sizeof(double), hipHostMallocDefault);

  hipMemcpyAsync(A_d, A, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  tt = omp_get_wtime();
  matrix_vector_kernel<<<N/BLOCK_SIZE+1,BLOCK_SIZE>>>(Ax_d, A_d, x_d, N);
  hipMemcpyAsync(Ax, Ax_d, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 2*N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double err = 0;
  for (long i = 0; i < N; i++) {
    err += (Ax_ref[i] - Ax[i]) * (Ax_ref[i] - Ax[i]);
  }
  printf("Error = %f\n", err);

  // Cleanup
  hipFree(A_d);
  hipFree(x_d);
  hipFree(Ax_d);
  hipHostFree(A);
  hipHostFree(x);
  hipHostFree(Ax);
  hipHostFree(Ax_ref);

  return 0;
}
