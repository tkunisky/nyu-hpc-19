
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

//
// General
//

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

// Reduction kernel for sum
__global__ void reduction_kernel(double* sum, const double* a, long N) {
	__shared__ double smem[BLOCK_SIZE];
	int idx = (blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx < N) smem[threadIdx.x] = a[idx];
	else smem[threadIdx.x] = 0;

	__syncthreads();
	if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
	__syncthreads();
	if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
	__syncthreads();
	if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
	__syncthreads();
	if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x + 64];
	__syncthreads();
	if (threadIdx.x <  32) {
		smem[threadIdx.x] += smem[threadIdx.x + 32];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 16];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 8];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 4];
		__syncwarp();
		smem[threadIdx.x] += smem[threadIdx.x + 2];
		__syncwarp();
		if (threadIdx.x == 0) {
			sum[blockIdx.x] = smem[0] + smem[1];
		}
	}
}

// CPU inner product implementation
void inner_product_ref(double* ip_ptr, const double* a, const double* b, long N) {
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i] * b[i];
  *ip_ptr = sum;
}

// Pointwise multiplication kernel
__global__ void pointwise_mult_kernel(double* xy, const double* x, const double* y, long N) {
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;
  if (idx < N) {
    xy[idx] = x[idx] * y[idx];
  }
}

// Wrapper for inner product kernel
void inner_product(double* ip, const double* x_d, const double* y_d, long N) {
  double *xy_d, *z_d;

  hipMalloc(&xy_d, N*sizeof(double));

  // Extra memory buffer for reduction across thread-blocks
  long N_work = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
       i > 1;
       i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) {
    N_work += i;
  }
  hipMalloc(&z_d, N_work*sizeof(double));

  pointwise_mult_kernel<<<N/BLOCK_SIZE+1,BLOCK_SIZE>>>(xy_d, x_d, y_d, N);

  double* ip_d = z_d;
  long Nb = (N + BLOCK_SIZE - 1) / (BLOCK_SIZE);
  reduction_kernel<<<Nb,BLOCK_SIZE>>>(ip_d, xy_d, N);
  while (Nb > 1) {
    long this_N = Nb;
    Nb = (Nb + BLOCK_SIZE - 1) / (BLOCK_SIZE);
    reduction_kernel<<<Nb,BLOCK_SIZE>>>(ip_d + this_N, ip_d, this_N);
    ip_d += this_N;
  }

  hipMemcpyAsync(ip, ip_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(xy_d);
  hipFree(z_d);
}

int main() {
  long N = (1UL<<25);

  double *x, *y, *x_d, *y_d;

  // Initialize vectors
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = drand48();
    y[i] = drand48();
  }

  // Get reference inner product
  double ip_ref, ip;
  double tt = omp_get_wtime();
  inner_product_ref(&ip_ref, x, y, N);
  printf("CPU Bandwidth = %f GB/s\n", 2*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  tt = omp_get_wtime();

  // Get GPU inner product
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&y_d, N*sizeof(double));

  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  tt = omp_get_wtime();
  inner_product(&ip, x_d, y_d, N);
  printf("GPU Bandwidth = %f GB/s\n", 2*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(ip - ip_ref));

  // Cleanup
  hipFree(x_d);
  hipFree(y_d);
  hipHostFree(x);
  hipHostFree(y);

  return 0;
}
